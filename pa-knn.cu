#include<stdio.h>
#include<hip/hip_runtime.h>
#include<time.h>
#include<math.h>
#define row 43500
#define col 10
#define test_row 14500
#define test_col 10


__global__

void KminNeighbourFind(double *distance1, int *d_kneighbours,int k,int set,int *res_class)
{
int i=blockDim.x*blockIdx.x+threadIdx.x;


int set_i;
 if(i<test_row)
{
for(int i1=0;i1<k;i1++)
{
	int min=2*(i1*test_row+i);
		for(int j1=i1+1;j1<row;j1++)
		{
			if(distance1[2*(j1*test_row+i)]<distance1[min])
			min=2*(j1*test_row+i);
		}

int dist=2*(i1*test_row+i),clas=2*(i1*test_row+i)+1;
double temp=distance1[dist];
distance1[dist]=distance1[min];
distance1[min]=temp;
//temp=distance1[clas];
//distance1[clas]=distance1[min+1];
//distance1[min+1]=temp;
int index= (int)distance1[min+1]-1;
 set_i=i*set;
index= index+set_i;
d_kneighbours[index]+=1;
//w=distnace1[2*(0*test_row+i)];
}
int max=0;
for(int l=1;l<set;l++)
{
if(d_kneighbours[set_i+l]>d_kneighbours[set_i+max])
max=l;
}
res_class[i]=max+1;

}

}
__global__
 
void Euclidian_distance(double *d_train,double *d_test, double *distance)
{
	int ro=blockIdx.x*blockDim.x+threadIdx.x;
        int co=blockIdx.y*blockDim.y+threadIdx.y;
       int distanceid=2*(ro*test_row+co);
     

	double sum=0,diff=0;
	//checking boundary condition
	if(ro<row && co<test_row)
	{
		for(int i=0; i<col-1; i++)
		{

			diff=(d_train[ro*col+i]-d_test[co*col+i]);
			sum+=diff*diff;
		}
		distance[distanceid]=sqrt(sum);
		distance[distanceid+1]=d_train[ro*col+col-1];
	}

// __syncthreads();
}
int main()
{
	
	 clock_t s_time,e_time;
    	double t_time;
    	FILE *myfile,*myfilet;
	int k,i,j;
	 double train[row*col],test1[test_row*test_col];
	double *d_train,*d_test;
	
       double *distance,*h_distance,*h_distance1;
 printf("Enter the k value to apply k nearest neighbour algorithm");
    scanf("%d",&k);
        
	 printf("\n");
 int set;
   printf("Enter the total classes present in your dataset\n");
   scanf("%d",&set);

	 myfile=fopen("shuttle.trn","r");
        if(myfile==NULL)
         {
           printf("data not open\n");
            exit(0);
         }
        else
         { 
	   printf("Successfully open\n");
	}
	
	myfilet=fopen("shuttle.tst","r");
	if(myfilet==NULL)
	{
	printf("Test data not open\n");
	exit(0);
	}
	else
	{
	printf("Test file open successfully\n");
	}

	
	//scanning train data
   	 for(i=0;i<row;i++)
   	 {
    	  for(j=0;j<col;j++)
    	       {
        	 fscanf(myfile,"%lf",&train[i*col+j]);
        	}
       	}

	//scanning test data

   	 for(i=0;i<test_row;i++)
    	{
     	 for(j=0;j<test_col;j++)
      	 {
        	 fscanf(myfilet,"%lf",&test1[i*test_col+j]);
      	 }
	    }

	hipError_t cudastatus ;
	cudastatus = hipDeviceReset () ;
	if(cudastatus!= hipSuccess)
	 {
	fprintf(stderr , " hipDeviceReset failed!" ) ;
	return 1;
	}
	cudastatus = hipSetDevice (0) ;
	if(cudastatus!=hipSuccess) 
	{
	fprintf(stderr , " hipSetDevice failed!");
	return 1;
	}
	else
	printf(" Working \n " ) ;


s_time=clock();

	
	size_t size=row*col*sizeof(double);
        size_t size1=test_row*test_col*sizeof(double);
	
	size_t distance_size=2*row*test_row*sizeof(double);
        size_t class_mem=test_row*sizeof(int);

 	int *res_class,*h_class;
	h_distance=(double*)malloc(distance_size);
	h_distance1=(double*)malloc(distance_size);
	h_class=(int*)malloc(class_mem);

	 //* Allocate matrices in device memory 
	   hipMalloc(&d_train, size);
      	hipMalloc(&d_test, size1);
	 hipMalloc(&distance,distance_size);
	hipMalloc(&res_class,class_mem);

	//copy the data from host to device memory
	hipMemcpy(d_train,train,size,hipMemcpyHostToDevice);
	  hipMemcpy(d_test,test1,size1,hipMemcpyHostToDevice);
   

	dim3 dimgrid((row-1)/16+1,(test_row-1)/16+1,1);
	dim3 dimblock(16,16,1);
	 Euclidian_distance<<<dimgrid,dimblock>>>(d_train,d_test,distance);
	hipMemcpy(h_distance,distance,distance_size,hipMemcpyDeviceToHost);


hipFree(d_train);
hipFree(d_test);

double *distance1;
//here code for min k neighbour cal
hipMalloc(&distance1,distance_size);
size_t neighbour_size =test_row*set*sizeof(int);
int *d_kneighbours;

hipMalloc(&d_kneighbours,neighbour_size);
hipMemcpy(distance,h_distance,distance_size,hipMemcpyHostToDevice);

int h_kneighbours[neighbour_size];
KminNeighbourFind<<<(test_row-1)/16+1,16>>>(distance,d_kneighbours,k,set,res_class);

hipMemcpy(h_distance1,distance,distance_size,hipMemcpyDeviceToHost);
hipMemcpy(h_kneighbours,d_kneighbours,neighbour_size,hipMemcpyDeviceToHost);
hipMemcpy(h_class,res_class,class_mem,hipMemcpyDeviceToHost);

hipFree(distance1);
hipFree(d_kneighbours);
hipFree(res_class);
/*for(i=0;i<test_row;i++)
{
for(j=0;j<set;j++)
{
printf("class freq of test case %d class no %d value %d\n",i+1,j,h_kneighbours[i*set+j]);
}
}
*/
int count=0;
for(i=0;i<test_row;i++)
{
if(test1[i*col+col-1]!=h_class[i])
count++;
printf("Given Test point %d  belongs to class %d\n",i+1,h_class[i]);
}
e_time=clock();
t_time=((double)(e_time-s_time))/1000000;
printf("Count unmachted %d",count);

printf("\n \n Total time taken %0.2lf second",t_time);

//cudaMemcpy(h_kneighbours,d_kneighbours,neighbour_size,cudaMemcpyDeviceToHost);
/*
for(i=0;i<row;i++)
{
for(j=0;j<1;j++)
{
printf("%lf %lf",h_distance[2*(i*test_row+j)],h_distance[2*(i*test_row+j)+1]);
}
printf("\n");
}

printf("K nearest one\n\n");
for(i=0;i<k;i++)
{
for(j=0;j<1;j++)
{
printf("%lf %lf",h_distance1[2*(i*test_row+j)],h_distance1[2*(i*test_row+j)+1]);
}
printf("\n");
}*/

return 0;
                      
	
}
